#include "hip/hip_runtime.h"
#include "CUDA_EXAMPLE/Helper.hpp"

#include <iostream>
#include <thrust/device_vector.h>

__global__ void addVecKernel(float *A, float *B, float* C) {
  int i = threadIdx.x;
  C[i] = A[i] + B[i];
}

// Assumes xs and ys are the same length
std::vector<float> addVec(const std::vector<float>& xs, const std::vector<float>& ys) {
  std::cout << "Using CUDA to add vectors...\n";
  std::vector<float> result;
  const auto size = xs.size();
  result.reserve(size);

  thrust::device_vector<float> A(xs.begin(), xs.end());
  thrust::device_vector<float> B(ys.begin(), ys.end());

  thrust::device_vector<float> C(size);

  addVecKernel<<<1, size>>>(
    thrust::raw_pointer_cast(A.data()),
    thrust::raw_pointer_cast(B.data()),
    thrust::raw_pointer_cast(C.data())
  );

  thrust::copy(C.begin(), C.end(), std::back_inserter(result));
  return result;
}
          
